#include "hip/hip_runtime.h"
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
extern "C" {
    #include "bmp.h"
}

typedef struct Color {
    unsigned int r, g, b;
} Color;

#define THREADS 1024

void CheckCudaError(char sms[], int line) {
  hipError_t error;
 
  error = hipGetLastError();
  if (error) {
    printf("(ERROR) %s - %s in %s at line %d\n", sms, hipGetErrorString(error), __FILE__, line);
    exit(EXIT_FAILURE);
  }
}

int square(int value) {
    return value * value;
}

void display_means(Color means[], int counts[], int N_colors) {
    int i;
    for (i = 0; i < N_colors; ++i) {
        fprintf(stderr, "mean %d:  ", i);
        fprintf(stderr, "r: %d, ", means[i].r);
        fprintf(stderr, "g: %d, ", means[i].g);
        fprintf(stderr, "b: %d, ", means[i].b);
        fprintf(stderr, "count: %d\n", counts[i]);
        
    }
    fprintf(stderr, "\n");
}

void display_assigns(int assigns[], int Size) {
    int i;
    for (i = 0; i < Size; ++i) {
        fprintf(stderr, "%d:  %d\n", i, assigns[i]);
    }
}

void init_means(Color means[], unsigned char *im, int Size_row, int N_colors, int Size) {
    int r;
    int i;
    for (i = 0; i < N_colors; ++i) {
        r = rand() % Size; 
        int index = (r*3/Size_row) * Size_row + ((r*3)%Size_row);
        means[i].r = im[index+2];
        means[i].g = im[index+1];
        means[i].b = im[index];
    }
}

void find_best_mean_seq(Color means[], int assigns[], unsigned char *im, int N, int ncolors, int Size_row) {
    int i;
    for (i = 0; i < N; ++i) {
        int j;
        int index = (i*3/Size_row) * Size_row + ((i*3)%Size_row);
        int dist_min = -1;
        int dist_act, assign;
        for (j = 0; j < ncolors; ++j) {
            dist_act = (im[index+2] - means[j].r)*(im[index+2] - means[j].r) + (im[index+1] - means[j].g)*(im[index+1] - means[j].g) + (im[index] - means[j].b)*(im[index] - means[j].b);
            if (dist_min == -1 || dist_act < dist_min) {
                dist_min = dist_act;
                assign = j;  
            }
        }
        assigns[i] = assign;
    }
}

__global__ void find_best_mean_par(Color means[], int assigns[], unsigned char *im, int N, int ncolors, int Size_row) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < N) {
        int j;
        int index = (id*3/Size_row) * Size_row + ((id*3)%Size_row);
        int dist_min = -1;
        int dist_act, assign;
        for (j = 0; j < ncolors; ++j) {
            dist_act = (im[index+2] - means[j].r)*(im[index+2] - means[j].r) + (im[index+1] - means[j].g)*(im[index+1] - means[j].g) + (im[index] - means[j].b)*(im[index] - means[j].b);
            if (dist_min == -1 || dist_act < dist_min) {
                dist_min = dist_act;
                assign = j;  
            }
        }
        assigns[id] = assign;
    }
}

void divide_sums_by_counts_seq(Color means_host[], int N_colors, Color new_means[], int counts[]) {
    int i;
    for (i = 0; i < N_colors; ++i) {
        //Turn 0/0 into 0/1 to avoid zero division.
        if(counts[i] == 0) counts[i] = 1;
        means_host[i].r = new_means[i].r / counts[i];
        means_host[i].g = new_means[i].g / counts[i];
        means_host[i].b = new_means[i].b / counts[i];
    }
}

__global__ void divide_sums_by_counts_par(Color means_device[], int N_colors, Color new_means[], int counts[]) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < N_colors) {
        //Turn 0/0 into 0/1 to avoid zero division.
        if(counts[id] == 0) counts[id] = 1;
        means_device[id].r = new_means[id].r / counts[id];
        means_device[id].g = new_means[id].g / counts[id];
        means_device[id].b = new_means[id].b / counts[id];
    }
}

void sum_up_and_count_points_seq(Color new_means[], int assigns[], unsigned char *im, int counts[], int Size_row, int Size) {
    int i;
    for (i = 0; i < Size; ++i) {
        int index = (i*3/Size_row) * Size_row + ((i*3)%Size_row);
        int imeans = assigns[i];
        new_means[imeans].r += im[index+2];
        new_means[imeans].g += im[index+1];
        new_means[imeans].b += im[index];
        counts[imeans] += 1;
    }
    
}

__global__ void matrix_reduction_color(Color new_means[], int assigns[], unsigned char *im, int Size_row, int Size, int N_colors, int offset) {
    extern __shared__ unsigned int shared[];
    
    unsigned int tid = threadIdx.x;
    unsigned int id = blockIdx.x*blockDim.x + threadIdx.x;
    
    //init shared
    for (int j = 0; j < N_colors; ++j) {
        
        if (j == assigns[id]) {
            int index = (id*3/Size_row) * Size_row + ((id*3)%Size_row);
            shared[tid*N_colors + j] = im[index+offset];
        }
        else {
            shared[tid*N_colors + j] = 0;
        }
    }
    
    __syncthreads();
    
    //reduccio
    unsigned int s;
    for(s=blockDim.x/2; s>0; s>>=1) { 
        if (tid < s) {
            for (int j = 0; j < N_colors; ++j) {
                shared[tid*N_colors + j] += shared[(tid + s)*N_colors + j];
            }
        }
        __syncthreads(); 
    }
    
    //copiar valors:
    if (tid == 0) {
       for (int j = 0; j < N_colors; ++j) {
            if (offset == 2) new_means[blockIdx.x*N_colors + j].r = shared[j];
            else if (offset == 1) new_means[blockIdx.x*N_colors + j].g = shared[j];
            else new_means[blockIdx.x*N_colors + j].b = shared[j];
            
        } 
    }
}

__global__ void matrix_reduction_color_2(Color new_means_2[], Color new_means[], int Size_row, int Size, int N_colors, int offset) {
    extern __shared__ unsigned int shared[];
    
    unsigned int tid = threadIdx.x;
    unsigned int id = blockIdx.x*(blockDim.x * 2) + threadIdx.x;
    
    //init shared
    for (int j = 0; j < N_colors; ++j) {
        
        if (offset == 2)         shared[tid*N_colors + j] = new_means[id*N_colors + j].r + new_means[blockDim.x*N_colors + id *N_colors + j].r;
        else if (offset == 1)    shared[tid*N_colors + j] = new_means[id*N_colors + j].g + new_means[blockDim.x*N_colors + id * N_colors + j].g;
        else                     shared[tid*N_colors + j] = new_means[id*N_colors + j].b + new_means[blockDim.x*N_colors + id *N_colors + j].b;
    }
    
   
    __syncthreads();
    
    //reduccio
    unsigned int s;
    for(s=blockDim.x/2; s>0; s>>=1) { 
        if (tid < s) {
            for (int j = 0; j < N_colors; ++j) {
                shared[tid*N_colors + j] += shared[(tid + s)*N_colors + j];
            }
        }
        __syncthreads(); 
    }
    
    //copiar valors:
    if (tid == 0) {
       for (int j = 0; j < N_colors; ++j) {
            if (offset == 2) new_means_2[blockIdx.x*N_colors + j].r = shared[j];
            else if (offset == 1) new_means_2[blockIdx.x*N_colors + j].g = shared[j];
            else new_means_2[blockIdx.x*N_colors + j].b = shared[j];
            
        } 
    }
}

__global__ void matrix_reduction_count(int counts[], int assigns[], unsigned char *im, int Size_row, int Size, int N_colors) {
    extern __shared__ unsigned int shared[];
    
    unsigned int tid = threadIdx.x;
    unsigned int id = blockIdx.x*blockDim.x + threadIdx.x;
    
    //init shared
    for (int j = 0; j < N_colors; ++j) {
        
        if (j == assigns[id]) {
            shared[tid*N_colors + j] = 1;
        }
        else {
            shared[tid*N_colors + j] = 0;
        }
    }
    __syncthreads();
    
    unsigned int s;
    for(s=blockDim.x/2; s>0; s>>=1) { 
        if (tid < s) {
            for (int j = 0; j < N_colors; ++j) {
                shared[tid*N_colors + j] += shared[(tid + s)*N_colors + j];
            }
        }
        __syncthreads(); 
    }
    
    //copiar valors:
    if (tid == 0) {
       for (int j = 0; j < N_colors; ++j) {
            counts[blockIdx.x*N_colors + j] = shared[j];
        } 
    }
}

__global__ void matrix_reduction_count_2(int counts_2[], int counts[], int Size_row, int Size, int N_colors) {
    extern __shared__ unsigned int shared[];
    
    unsigned int tid = threadIdx.x;
    unsigned int id = blockIdx.x*(blockDim.x * 2) + threadIdx.x;
    
    //init shared
    for (int j = 0; j < N_colors; ++j) {
        
        shared[tid*N_colors + j] = counts[id*N_colors + j] + counts[blockDim.x*N_colors + (id * N_colors) + j];
    }
    
    __syncthreads();
    
    //reduccio
    unsigned int s;
    for(s=blockDim.x/2; s>0; s>>=1) { 
        if (tid < s) {
            for (int j = 0; j < N_colors; ++j) {
                shared[tid*N_colors + j] += shared[(tid + s)*N_colors + j];
            }
        }
        __syncthreads(); 
    }
    
    //copiar valors:
    if (tid == 0) {
       for (int j = 0; j < N_colors; ++j) {
            counts_2[blockIdx.x*N_colors + j] = shared[j];
        } 
    }
}

__global__ void sum_up_and_count_points_par(Color new_means[], int assigns[], unsigned char *im, int counts[],
            int Size_row, int Size, int N_colors, int s_counts[], Color s_new_means[]) {

    unsigned int tid = threadIdx.x;
    unsigned int id = blockIdx.x*blockDim.x + threadIdx.x;
    
    //inicialitzar 
    for (int j = 0; j < N_colors; ++j) {
        
        if (j == assigns[id]) {
            int index = (id*3/Size_row) * Size_row + ((id*3)%Size_row);
            s_new_means[blockIdx.x*blockDim.x + tid*N_colors + j].r = im[index+2];
            s_new_means[blockIdx.x*blockDim.x + tid*N_colors + j].g = im[index+1];
            s_new_means[blockIdx.x*blockDim.x + tid*N_colors + j].b = im[index];
            s_counts[blockIdx.x*blockDim.x + tid*N_colors + j] = 1;
        }
        else {
            s_new_means[blockIdx.x*blockDim.x + tid*N_colors + j].r = 0;
            s_new_means[blockIdx.x*blockDim.x + tid*N_colors + j].g = 0;
            s_new_means[blockIdx.x*blockDim.x + tid*N_colors + j].b = 0;
            s_counts[blockIdx.x*blockDim.x + tid*N_colors + j] = 0;
        }
    }
    __syncthreads();
    
    //reduccio
    unsigned int s;
    for(s=1; s < blockDim.x; s *= 2) { 
        if (tid % (2*s) == 0) {
            for (int j = 0; j < N_colors; ++j) {
                
                s_new_means[blockIdx.x*blockDim.x + tid*N_colors + j].r += s_new_means[(tid + s)*N_colors + j].r;
                s_new_means[blockIdx.x*blockDim.x + tid*N_colors + j].g += s_new_means[(tid + s)*N_colors + j].g;
                s_new_means[blockIdx.x*blockDim.x + tid*N_colors + j].b += s_new_means[(tid + s)*N_colors + j].b;
                
                s_counts[blockIdx.x*blockDim.x + tid*N_colors + j] += s_counts[(tid + s)*N_colors + j];
            }
        }
        __syncthreads(); 
    }
    __syncthreads();
    //copiar valors:
    if (tid == 0) {
       for (int j = 0; j < N_colors; ++j) {
            new_means[blockIdx.x*N_colors + j].r = s_new_means[j].r;
            new_means[blockIdx.x*N_colors + j].g = s_new_means[j].g;
            new_means[blockIdx.x*N_colors + j].b = s_new_means[j].b;
            counts[j] = s_counts[j];
        } 
    }
}

__global__ void findandsum(Color means[],Color new_means[], int assigns[], unsigned char *im, int counts[],
    int Size_row, int Size, int ncolors) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < Size) {
		int j;
        int index = (id*3/Size_row) * Size_row + ((id*3)%Size_row);
        int dist_min = -1;
        int dist_act, assign;
        for (j = 0; j < ncolors; ++j) {
            dist_act = (im[index+2] - means[j].r)*(im[index+2] - means[j].r) + (im[index+1] - means[j].g)*(im[index+1] - means[j].g) + (im[index] - means[j].b)*(im[index] - means[j].b);
            if (dist_min == -1 || dist_act < dist_min) {
                dist_min = dist_act;
                assign = j;  
            }
        }
        assigns[id] = assign;
        
        atomicAdd(&new_means[assign].r, im[index+2]);
        atomicAdd(&new_means[assign].g, im[index+1]);
        atomicAdd(&new_means[assign].b, im[index]);
        atomicAdd(&counts[assign], 1);
    }
    
}

void assign_colors_seq(Color means[], int assigns[], unsigned char *im, int Size_row, int Size) {
    int i;
    for (i = 0; i < Size; ++i) {
        int index = (i*3/Size_row) * Size_row + ((i*3)%Size_row);
        im[index]=means[assigns[i]].b;
        im[index + 1]=means[assigns[i]].g;
        im[index + 2]=means[assigns[i]].r;  
    }
}

__global__ void assign_colors_par(Color means[], int assigns[], unsigned char *im, int Size_row, int Size) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < Size) {
        int index = (id*3/Size_row) * Size_row + ((id*3)%Size_row);
        im[index]=means[assigns[id]].b;
        im[index + 1]=means[assigns[id]].g;
        im[index + 2]=means[assigns[id]].r;  
    }
}

int main(int c, char *v[])
{
    int N_colors;
    if (c < 4 || c > 5) {
        fprintf(stderr, "usage: %s ppm_file n_iterations seed n_colors\n", v[0]);
        return -1;
    }
    
    else if (c == 4) N_colors = 16;
    else if (c == 5) N_colors = atoi(v[4]) ? : 16;
    
    //read image:
    bmpInfoHeader infoHeader;
    unsigned char *im_host = LoadBMP(v[1], &infoHeader);
    
    //init variables:
    float elapsedTime;
    int N_iterations = atoi(v[2]);
    int Size_row = ((infoHeader.width*3 + 3) / 4) * 4;
    int width = infoHeader.width;
    int height = infoHeader.height;
    int Size = width * height;
    
    //init seed
    srand(atoi(v[3]));
    
    //init grid, block, nThreads:
    unsigned int nBlocks, nBlocksMeans, nThreads;
    nThreads = THREADS;
    nBlocks = (Size + nThreads - 1)/nThreads;

    dim3 dimGrid(nBlocks, 1, 1);
    dim3 dimBlock(nThreads, 1, 1);
    
    nBlocksMeans = (N_colors + nThreads - 1)/nThreads;

    dim3 dimGridMeans(nBlocksMeans, 1, 1); 

    //obtenir memoria HOST:
    Color *means_host;
    means_host = (Color*) malloc(N_colors*sizeof(Color));
    int *counts_host;
    counts_host = (int*) malloc(sizeof(int) * N_colors);
    
    Color *means_host_red;
    means_host_red = (Color*) malloc((nBlocks/(2*nThreads)) * N_colors*sizeof(Color));
    int *counts_host_red;
    counts_host_red = (int*) malloc((nBlocks/(2*nThreads)) * sizeof(int) * N_colors);
    
    
    //inicialitzar means:
    init_means(means_host, im_host, Size_row, N_colors, Size);
    
    //cuda events:
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    

    //obtenir memoria DEVICE:
    Color *means_device;
    Color *new_means;
    int *counts;
    Color *new_means_2;
    int *counts_2;
    
    int *assigns;
    unsigned char *im_device;
    
    hipMalloc((Color**)&means_device, N_colors*sizeof(Color));
    
    hipMalloc((Color**)&new_means, nBlocks * N_colors*sizeof(Color));
    hipMalloc((int**)&counts, nBlocks * N_colors * sizeof (int));
    
    hipMalloc((Color**)&new_means_2, (nBlocks/(2*nThreads)) * N_colors*sizeof(Color));
    hipMalloc((int**)&counts_2, (nBlocks/(2*nThreads)) * N_colors * sizeof (int));
    
    hipMalloc((int**)&assigns, Size*sizeof(int));
    hipMalloc((unsigned char**)&im_device, infoHeader.imgsize* sizeof(unsigned char));
    CheckCudaError((char *) "Obtener Memoria en el device", __LINE__);
    
    //copiar dades al device:
    hipMemcpy(im_device, im_host, infoHeader.imgsize * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(means_device, means_host, N_colors*sizeof(Color), hipMemcpyHostToDevice);
    CheckCudaError((char *) "Copiar Datos Host --> Device", __LINE__);

    
    int shared_memory_size = N_colors*THREADS * sizeof(unsigned int);
    
    //START RECORD!!
    hipEventRecord(start, 0);
    
    //executem k means:
    int it;
    for (it = 0; it < N_iterations; ++it) {
        
        //set counts and new_means to 0
        hipMemset (counts, 0, nBlocks * sizeof (int) * N_colors);
        hipMemset (new_means, 0, nBlocks * sizeof (Color) * N_colors);
        
        //for each pixel find the best mean.
        find_best_mean_par<<<dimGrid, dimBlock>>>(means_device, assigns, im_device, Size, N_colors, Size_row);
        
        hipDeviceSynchronize();
        
        /*
        //Sum up and count points for each cluster.
        sum_up_and_count_points_par<<<dimGrid, dimBlock>>>(new_means, assigns, im_device, counts, Size_row, Size, N_colors, s_counts, s_new_means);
        hipDeviceSynchronize();
        */
        
        
        matrix_reduction_count<<<dimGrid, dimBlock, shared_memory_size>>>(counts, assigns, im_device, Size_row, Size, N_colors);
        matrix_reduction_color<<<dimGrid, dimBlock, shared_memory_size>>>(new_means, assigns, im_device, Size_row, Size, N_colors, 2);
        matrix_reduction_color<<<dimGrid, dimBlock, shared_memory_size>>>(new_means, assigns, im_device, Size_row, Size, N_colors, 1);
        matrix_reduction_color<<<dimGrid, dimBlock, shared_memory_size>>>(new_means, assigns, im_device, Size_row, Size, N_colors, 0);
        
	hipDeviceSynchronize();
        //volmemos a hacer otra reduccion
        matrix_reduction_count_2<<<nBlocks/(2*nThreads), dimBlock, shared_memory_size>>>(counts_2, counts, Size_row, Size, N_colors);
        matrix_reduction_color_2<<<nBlocks/(2*nThreads), dimBlock, shared_memory_size>>>(new_means_2, new_means, Size_row, Size, N_colors, 2);
        matrix_reduction_color_2<<<nBlocks/(2*nThreads), dimBlock, shared_memory_size>>>(new_means_2, new_means, Size_row, Size, N_colors, 1);
        matrix_reduction_color_2<<<nBlocks/(2*nThreads), dimBlock, shared_memory_size>>>(new_means_2, new_means, Size_row, Size, N_colors, 0);

        hipDeviceSynchronize();
        
        
        hipMemcpy(means_host_red, new_means_2, (nBlocks/(2*nThreads)) * N_colors * sizeof(Color), hipMemcpyDeviceToHost);
        hipMemcpy(counts_host_red, counts_2, (nBlocks/(2*nThreads)) * N_colors * sizeof(int), hipMemcpyDeviceToHost);
        
        memset(counts_host, 0, sizeof (int) * N_colors);
        memset(means_host, 0, sizeof (Color) * N_colors);
        
        int i, j;
        for (i = 0; i < nBlocks/(2*nThreads); ++i) {
            for (j = 0; j < N_colors; ++j) {
                counts_host[j] += counts_host_red[i*N_colors + j];
                means_host[j].r += means_host_red[i*N_colors + j].r;
                means_host[j].g += means_host_red[i*N_colors + j].g;
                means_host[j].b += means_host_red[i*N_colors + j].b;
            }
        }
        
        //aqui tenemos los vectores finales ya reducidos
        hipMemcpy(new_means, means_host, N_colors * sizeof(Color), hipMemcpyHostToDevice);
        hipMemcpy(counts, counts_host, N_colors * sizeof(int), hipMemcpyHostToDevice);
        
        
        /*
		findandsum<<<dimGrid, dimBlock>>>(means_device,new_means, assigns, im_device, counts, Size_row, Size, N_colors);
		hipDeviceSynchronize();
        */
        
        
    
        
        //Divide sums by counts to get new centroids.
        divide_sums_by_counts_par<<<dimGridMeans, dimBlock>>>(means_device, N_colors, new_means, counts);
        
        hipDeviceSynchronize();
        
        
        
    }
    
    //assignem colors:
    assign_colors_par<<<dimGrid, dimBlock>>>(means_device, assigns, im_device, Size_row, Size);
    
    //copy to host:
    hipMemcpy(im_host, im_device, infoHeader.imgsize * sizeof(unsigned char), hipMemcpyDeviceToHost);
    
    //STOP RECORD!!
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    

    
    //save image
    SaveBMP("sortida.bmp", &infoHeader, im_host);
	
    DisplayInfo("sortida.bmp", &infoHeader);
    
    int bytes_read_written = 2 * infoHeader.imgsize* sizeof(unsigned char) + //leer imagen y copiarla
                             N_iterations * (                                //en cada iteracion se hace:
                                sizeof (int) * 2 * N_colors +                   //leer y modificar counts
                                sizeof (Color) * N_colors +                     //leer y modificar medias
                                Size * 2 * sizeof(int) +                        //leer y modificar las asignaciones
                                Size * 3 * sizeof (unsigned char)               //leer datos de imagen
                             );       
    
    printf("\Quantization CUDA\n");
    printf("Image Size: %d\n", Size);
    printf("nThreads: %d\n", nThreads);
    printf("nBlocks: %d\n", nBlocks);
    printf("Tiempo Total Versio 4 = %4.6f ms\n", elapsedTime);

    hipEventDestroy(start);
    hipEventDestroy(stop);    
    
    //alliberar memoria HOST:
    free(im_host);
    free(means_host);
    
    //alliberar memoria DEVICE:
    hipFree(means_device);
    hipFree(new_means);
    hipFree(new_means_2);
    hipFree(assigns);
    hipFree(im_device);
    hipFree(counts);
    hipFree(counts_2);
    return 0;
}
